#include "hip/hip_runtime.h"
#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define N 20
#define MAXCOL 10000
#define NL_min 0
#define NL_max 0
#define NL_step 1
#define Ng_max 1
#define N_THREADS_PER_BLOCK 1024
#define N_BLOCKS_X 65535
#define N_BLOCKS_Y 65535
#define N_BLOCKS_z 65535

typedef struct {
  int All_sync_count1[NL_max-NL_min+1][Ng_max];
  int All_sync_count2[NL_max-NL_min+1];
  double dt, epsilon, vth, vreset, a, b, tol;
  long Nstep;
} global_mem;

typedef struct {
  int iL, nL_break, ig, ic;
  double w[N][N];
} simulation_params;

typedef struct {
  int ic, iL, nL_break, ig;
  double v_init[N];
  double tspike[N][MAXCOL];
} simulation_result;

/* creates a network (adj. matrix) of N neurons in "w" with "nL" synapses missing */
void synaptic_weighs_connected_network(double w[][N], int nL);

__global__ void simulate(simulation_params *params, simulation_result *results, global_mem *g_mem) {
  // "threadId" is used as an index into the arrays "params" and "results".
  // Everything that was being written to a file is now returned in a struct.
  int blockId = blockIdx.z * gridDim.x * gridDim.y + 
                blockIdx.y * gridDim.x + 
                blockIdx.x;
  int threadId = blockId * blockDim.x + threadIdx.x;
  
  results[threadId].ic = params[threadId].ic;
  results[threadId].iL = params[threadId].iL;
  results[threadId].nL_break = params[threadId].nL_break;
  results[threadId].ig = params[threadId].ig;
  
  int i, k kk, t_old, t_new, InSync_neurons;
  int spike_count[N], spike[N], push_up_flag[N];
  double f0, f1, f2, f3, f4, tspike_diff1, tspike_diff2;
  double v_old[N], v_new[N], push_up_amnt[N];
  double v_initnew[20]= {0.00778832, 0.355919, 0.426307, 0.183062,
                         0.272762, 0.532633, 0.339171, 0.242097,
                         0.523038, 0.638838, 0.632368, 0.778564,
                         0.110892, 0.347691, 0.696286, 0.791943,
                         0.5257, 0.127494, 0.716965, 0.151006};

  // Generate initial state
  for(kk = 0; kk < N; kk++) {
    /* Change rand() to cuRAND:: */
    results[threadId].v_init[kk] = rand() % (g_mem->vth*1000);
    results[threadId].v_init[kk] = results[threadId].v_init[kk] / 100000;
    v_old[kk] = results[threadId].v_init[kk];
  }
  
  for(kk = 0; kk < N; kk++){		
    results[threadId].v_init[kk] = v_initnew[kk];
    v_old[kk] = results[threadId].v_init[kk];
  }

  // initialize arrays
  for(k=0; k < N; k++) {
    spike_count[k] = 0; //keeps a count of the number spikes in neuron k so far
  }

  for(k = 0; k < N; k++){
    for(i = 0; i < MAXCOL; i++){
      results[threadId].tspike[k][i] = 0; // counts the spike time of "i_th" spike of neuron number "k"
    }
  }

  // Time loop begins
  t_old = 0;
  for(i = 1; i < g_mem->Nstep; i++) { 	

    t_new = i*(g_mem->dt);

    // Identify (1) the neurons that spiked in previous time step, (2) time of all the spikes of each neuron
    // (3) total number of spikes in each neuron so far
    for(kk = 0; kk < N; kk++) {
      if(v_old[kk] >= g_mem->vth) {
        spike[kk] = 1; // if neuron spiked
        spike_count[kk]++ ;
        results[threadId].tspike[kk][spike_count[kk]] = t_old;
      }
      else {	
        spike[kk] = 0; // if neuron did not spike
      }
    }

    // Find voltage push-up amount for each neuron (if atleast one neuron other than itself spiked)
    for(kk = 0; kk < N; kk++) {
      push_up_amnt[kk] = 0; // initialize these arrays at every time step
      push_up_flag[kk] = 0;
    }
    for(kk = 0; kk < N; kk++) {
      for(k = 0; k < N; k++) {	
        if(k != kk && spike[kk] != 1 && spike[k]==1) {
          push_up_amnt[kk] = push_up_amnt[kk] +
                             (g_mem->epsilon) * params.w[kk][k] * spike[k];
          push_up_flag[kk] = 1;
        }
      }
    }

    // Finally update voltages of each neuron - using Euler method if no neuron fired & by pushing up the
    // voltage value by push_up_amnt if some neurons fired.
    for(kk = 0; kk < N; kk++) {
      if(v_old[kk] < g_mem->vth) { 
        if(push_up_flag[kk] == 1) {

          v_new[kk] = v_old[kk] + push_up_amnt[kk];
          
          if(v_new[kk] >= g_mem->vth) {
            v_new[kk] = g_mem->vreset;
            spike_count[kk]++;
            results[threadId].tspike[kk][spike_count[kk]] = t_old;
          }

        }
        else if(push_up_flag[kk] == 0) {
          f0 = g_mem->a - g_mem->b * v_old[kk];
          f1 = g_mem->a - g_mem->b * (v_old[kk] + f0 * 0.5 * g_mem->dt);
          f2 = g_mem->a - g_mem->b * (v_old[kk] + f1 * 0.5 * g_mem->dt);
          f3 = g_mem->a - g_mem->b * (v_old[kk] + f2 * g_mem->dt);
          v_new[kk] = v_old[kk] + g_mem->dt * (f0 + 2 * f1 + 2 * f2 + f3) / 6;
        }
      }
      else if (v_old[kk] >= g_mem->vth) {
        v_new[kk] = g_mem->vreset;
      }
    }
    
    // swap v_old & v_new for next time iteration
    for(kk = 0; kk < N; kk++) {
      v_old[kk] = v_new[kk];
    }

    // Advance time
    t_old = t_new;

  } // Time loop ends

  // Count number of iL-networks where all neurons fire in sync

  InSync_neurons = 1;
  for(kk = 1; kk < N; kk++) {
    // TOASK: What are these "10" and "11"?
    tspike_diff1 = fabs(results[threadId].tspike[0][spike_count[0] - 11] -
                        results[threadId].tspike[kk][spike_count[kk] - 11]);
    tspike_diff2 = fabs(results[threadId].tspike[0][spike_count[0] - 10] -
                        results[threadId].tspike[kk][spike_count[kk] - 10]);
    if(tspike_diff1 < g_mem->tol && tspike_diff2 < g_mem->tol) {
      InSync_neurons++; // count number of neurons firing in sync for the chosen initial condition
    }
  }
  if(InSync_neurons == N) {
    //g_mem->All_sync_count1[params[threadId].iL][params[threadId].ig]++; // count number of ic's that yield All-sync for iL-iG network.
    g_mem->All_sync_count2[params[threadId].iL]++;
    //printf("Number of instances of full sync = %d \n",All_sync_count2[iL]);
    //fprintf(all_sync,"Number of instances of full sync = %d \n",All_sync_count2[0]);
  }

  // TOASK: What is happening here?
  // Write spike time on file
  for(kk=0;kk<N;kk++) {
    tmp1 = 10000*results[threadId].tspike[kk][spike_count[kk]-7];
    tmp2 = 10000*results[threadId].tspike[kk][spike_count[kk]-8];
    tmp3 = 10000*results[threadId].tspike[kk][spike_count[kk]-9];
    tmp4 = 10000*results[threadId].tspike[kk][spike_count[kk]-10];
    tmp5 = 10000*results[threadId].tspike[kk][spike_count[kk]-11];
    tmp6 = 10000*results[threadId].tspike[kk][spike_count[kk]-12];
    tmp7 = 10000*results[threadId].tspike[kk][spike_count[kk]-13];
  //fprintf(spike_time,"%d \t %lu \t %lu \t %lu \t %lu \t %lu \t \%d \n",kk,tmp1,tmp2,tmp3,tmp4,tmp5,flag_unconnctd_graph);
                      //fprintf(spike_time,"%d \t %lu \t %lu \t %lu \t %lu \t %lu \t %lu \t %lu \n",kk,tmp1,tmp2,tmp3,tmp4,tmp5,tmp6,tmp7);
  }
}

void synaptic_weighs_connected_network(double w[][N], int nL) {
  int i, j, num_removed;
	int degree[N]; // degree : number of neurons this neuron has synapses with
	// Create a completely connected network
	for(i = 0; i < N; ++i) {
		for(j = 0; j < N; ++j) {
			w[i][j] = 1;
		}
	}
	// Remove self-connections
	for(int i = 0; i < N; ++i) {
		w[i][i] = 0;
		degree[i] = N-1;
	}
	// Keep removing synapses till nL aren't removed
	num_removed = 0;
	while(num_removed < nL) {
		i = rand() % N;
		j = rand() % N;
		// If there is a synapse between neurons i & j and connectivity can be ensured, remove it
		if(w[i][j] != 0 && degree[i] > 1 && degree[j] > 1) {
			w[i][j] = 0;
			w[j][i] = 0;
			--degree[i];
			--degree[j];
			++num_removed;
		}
	}
}

int main() {
  
  return 0;
}