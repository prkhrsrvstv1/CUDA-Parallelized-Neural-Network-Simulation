#include "hip/hip_runtime.h"
#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define N 2
#define MAXCOL 10000
#define Nic 1
#define NL_min 0
#define NL_max 0
#define NL_step 1
#define Ng_max 1
#define N_THREADS_PER_BLOCK 1
#define N_BLOCKS 1

typedef struct {
  int All_sync_count1[NL_max-NL_min+1][Ng_max];
  int All_sync_count2[NL_max-NL_min+1];
  double dt, epsilon, vth, vreset, a, b, tol;
  long Nstep;
} global_mem;

typedef struct {
  int iL, ig, ic;
} simulation_params;

typedef struct {
  int ic, iL, nL_break, ig;
  double v_init[N];
  double tspike[N][MAXCOL];
} simulation_result;

/* Generate a adjacency matrix for a coonnected graph with nL edges missing */
__device__ int synaptic_weights_connected_network(double w[][N], int nL) {

  int i,j,k,kk,neuron1,neuron2;
  double w_flag[N][N];
  int syn_to_remove, tot_syn_removed ;
  int connected_nodes[N] ;
  int current_ptr, endptr, parent_node;
  int flag_connected = 0 ;
  int flag_already_connected;

  // GENERATE AN ALL-TO-ALL NETWORK ************************************************************************
  for(i = 0; i < N; i++) {
    for(j = 0; j < N; j++) {
      if(j != i){
        w[i][j] = 1;
      }
      else if(j == i){
        w[i][j] =0;
      } 
    }
  }

  // REMOVE SYNAPSES FROM ABOVE ALL-TO-ALL NETWORK *********************************************************

  syn_to_remove = nL;
  tot_syn_removed = 0;

  // Initialize array w_flag
  for(k = 0; k < N; k++) {
    for(kk = 0; kk < N; kk++) {
      w_flag[k][kk] = 0; // w_flag[k][kk] is changed to value 1, if the synapse between k --> kk is removed
    }
  }

  // Generate a new network by removing synapses randomly
  while(tot_syn_removed < syn_to_remove) {
    int neuron1 = rand() % N;
    int neuron2 = rand() % N;
    if(neuron1 != neuron2) {
      if(w_flag[neuron1][neuron2] == 0) { // synapse between these two neurons has not been changed.
        w_flag[neuron1][neuron2] = 1;
        w_flag[neuron2][neuron1] = 1;
        w[neuron1][neuron2] = 0;
        w[neuron2][neuron1] = w[neuron1][neuron2];
        tot_syn_removed++;
      }
    }
  }


  // Is the network generated above connected ? /////////////


  //w[0][0] = 0; w[0][1] = 1; w[0][2] = 1; w[0][3] = 0; w[0][4] = 1; w[0][5] = 0;

  //w[1][0] = w[0][1]; w[1][1] = 0 ; w[1][2] = 1 ; w[1][3] = 0; w[1][4] = 0; w[1][5] = 1;

  //w[2][0] = w[0][2]; w[2][1] = w[1][2] ; w[2][2] = 0 ; w[2][3] = 0; w[2][4] = 1; w[2][5] = 0;

  //w[3][0] = w[0][3]; w[3][1] = w[1][3] ; w[3][2] = w[2][3] ; w[3][3] = 0; w[3][4] = 0; w[3][5] = 0;

  //w[4][0] = w[0][4]; w[4][1] = w[1][4] ; w[4][2] = w[2][4] ; w[4][3] = w[3][4]; w[4][4] = 0; w[4][5] = 1;

  //w[5][0] = w[0][5]; w[5][1] = w[1][5] ; w[5][2] = w[2][5] ; w[5][3] = w[3][5]; w[5][4] = w[4][5]; w[5][5] = 0;

  //w[0][0] = 0 ; w[0][1] = 0; w[0][2] = 1; w[0][3]=0;
  //w[1][0] = w[0][1] ; w[1][1] = 0;  w[1][2] = 1; w[1][3] =0;
  //w[2][0]=w[0][2] ; w[2][1]=w[1][2]; w[2][2] =0; w[2][3] = 1;
  //w[3][0] = w[0][3] ; w[3][1] = w[1][3] ; w[3][2] = w[2][3] ; w[3][3]=0 ;

  for(k = 0; k < N; k++) {
    for(kk = 0; kk < N; kk++) {
      w_flag[k][kk] = 0; // w_flag[k][kk] is changed to value 1, if the synapse between k --> kk is removed
    }
  }

  connected_nodes[0] = 0;
  for(i=1;i<N;i++) {
    connected_nodes[i] = -1;
  }
  current_ptr = 0;
  endptr = 0 ;  // points towards the last non-zero element in the connected_nodes array

  while(current_ptr <= endptr) {

    for(i = 0; i < N; i++) {
      parent_node = connected_nodes[current_ptr] ;

      flag_already_connected = 0 ;

      for(j = 0; j <= endptr; j++) {
        if(connected_nodes[j] == i) {
          flag_already_connected = 1;
        }
      }

      if(w[parent_node][i] == 1) {
        if(w_flag[parent_node][i] == 0) {
          if(flag_already_connected ==0) {
            endptr ++ ;
            connected_nodes[endptr] = i ; // stores node numbers connected to parent_node

            w_flag[parent_node][i] = 1 ;
            w_flag[i][parent_node] = w_flag[parent_node][i] ; //links already visited

            //printf("i= %d \t endptr= %d \t current_ptr= %d \t connected_nodes[endptr] = %d \n",i, endptr,current_ptr,connected_nodes[endptr]);
          }
        }
      }

      if (i == N-1) {
        current_ptr++ ;
      }	
    }
  }

  if(endptr == N-1) {
    flag_connected = 1 ;
  }

  return flag_connected;
}

/* Create weight matrices in GPU memory */
__global__ void store_weights(double w[(NL_max - NL_min) * Ng_max / NL_step][N][N]) {
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  int nL_break = NL_min + threadId * NL_step;
  int flag_connected;
  for(int i = 0; i < Ng_max; ++i) {
    flag_connected = 0;
    do {
      flag_connected = synaptic_weights_connected_network(w[threadId * Ng_max + i], nL_break);
    } while(flag_connected == 0);
  }
}

/* Run a simulation on a single thread */
__global__ void simulate(simulation_params *params, simulation_result *results, global_mem *g_mem, double w[(NL_max - NL_min) * Ng_max / NL_step][N][N]) {
  // "threadId" is used as an index into the arrays "params" and "results".
  // Everything that was being written to a file is now returned in a struct.
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  
  results[threadId].ic = params[threadId].ic;
  results[threadId].iL = params[threadId].iL;
  results[threadId].nL_break = params[threadId].nL_break;
  results[threadId].ig = params[threadId].ig;
  
  int i, k kk, t_old, t_new, InSync_neurons;
  int spike_count[N], spike[N], push_up_flag[N];
  double f0, f1, f2, f3, f4, tspike_diff1, tspike_diff2;
  double v_old[N], v_new[N], push_up_amnt[N];
  double v_initnew[20]= {0.00778832, 0.355919, 0.426307, 0.183062,
                         0.272762, 0.532633, 0.339171, 0.242097,
                         0.523038, 0.638838, 0.632368, 0.778564,
                         0.110892, 0.347691, 0.696286, 0.791943,
                         0.5257, 0.127494, 0.716965, 0.151006};

  // Generate initial state
  for(kk = 0; kk < N; kk++) {
    /* Change rand() to cuRAND:: */
    results[threadId].v_init[kk] = rand() % (g_mem->vth*1000);
    results[threadId].v_init[kk] = results[threadId].v_init[kk] / 1000;
    v_old[kk] = results[threadId].v_init[kk];
  }
  
  for(kk = 0; kk < N; kk++){		
    results[threadId].v_init[kk] = v_initnew[kk];
    v_old[kk] = results[threadId].v_init[kk];
  }

  // initialize arrays
  for(k=0; k < N; k++) {
    spike_count[k] = 0; //keeps a count of the number spikes in neuron k so far
  }

  for(k = 0; k < N; k++){
    for(i = 0; i < MAXCOL; i++){
      results[threadId].tspike[k][i] = 0; // counts the spike time of "i_th" spike of neuron number "k"
    }
  }

  // Time loop begins
  t_old = 0;
  for(i = 1; i < g_mem->Nstep; i++) { 	

    t_new = i*(g_mem->dt);

    // Identify (1) the neurons that spiked in previous time step, (2) time of all the spikes of each neuron
    // (3) total number of spikes in each neuron so far
    for(kk = 0; kk < N; kk++) {
      if(v_old[kk] >= g_mem->vth) {
        spike[kk] = 1; // if neuron spiked
        spike_count[kk]++ ;
        results[threadId].tspike[kk][spike_count[kk]] = t_old;
      }
      else {	
        spike[kk] = 0; // if neuron did not spike
      }
    }

    // Find voltage push-up amount for each neuron (if atleast one neuron other than itself spiked)
    for(kk = 0; kk < N; kk++) {
      push_up_amnt[kk] = 0; // initialize these arrays at every time step
      push_up_flag[kk] = 0;
    }
    for(kk = 0; kk < N; kk++) {
      for(k = 0; k < N; k++) {	
        if(k != kk && spike[kk] != 1 && spike[k]==1) {
          push_up_amnt[kk] = push_up_amnt[kk] +
                             (g_mem->epsilon) * w[threadId % Nic][kk][k] * spike[k];
          push_up_flag[kk] = 1;
        }
      }
    }

    // Finally update voltages of each neuron - using Euler method if no neuron fired & by pushing up the
    // voltage value by push_up_amnt if some neurons fired.
    for(kk = 0; kk < N; kk++) {
      if(v_old[kk] < g_mem->vth) { 
        if(push_up_flag[kk] == 1) {

          v_new[kk] = v_old[kk] + push_up_amnt[kk];
          
          if(v_new[kk] >= g_mem->vth) {
            v_new[kk] = g_mem->vreset;
            spike_count[kk]++;
            results[threadId].tspike[kk][spike_count[kk]] = t_old;
          }

        }
        else if(push_up_flag[kk] == 0) {
          f0 = g_mem->a - g_mem->b * v_old[kk];
          f1 = g_mem->a - g_mem->b * (v_old[kk] + f0 * 0.5 * g_mem->dt);
          f2 = g_mem->a - g_mem->b * (v_old[kk] + f1 * 0.5 * g_mem->dt);
          f3 = g_mem->a - g_mem->b * (v_old[kk] + f2 * g_mem->dt);
          v_new[kk] = v_old[kk] + g_mem->dt * (f0 + 2 * f1 + 2 * f2 + f3) / 6;
        }
      }
      else if (v_old[kk] >= g_mem->vth) {
        v_new[kk] = g_mem->vreset;
      }
    }
    
    // swap v_old & v_new for next time iteration
    for(kk = 0; kk < N; kk++) {
      v_old[kk] = v_new[kk];
    }

    // Advance time
    t_old = t_new;

  } // Time loop ends

  // Count number of iL-networks where all neurons fire in sync

  InSync_neurons = 1;
  for(kk = 1; kk < N; kk++) {
    // TOASK: What are these "10" and "11"?
    tspike_diff1 = fabs(results[threadId].tspike[0][spike_count[0] - 11] -
                        results[threadId].tspike[kk][spike_count[kk] - 11]);
    tspike_diff2 = fabs(results[threadId].tspike[0][spike_count[0] - 10] -
                        results[threadId].tspike[kk][spike_count[kk] - 10]);
    if(tspike_diff1 < g_mem->tol && tspike_diff2 < g_mem->tol) {
      InSync_neurons++; // count number of neurons firing in sync for the chosen initial condition
    }
  }
  if(InSync_neurons == N) {
    //g_mem->All_sync_count1[params[threadId].iL][params[threadId].ig]++; // count number of ic's that yield All-sync for iL-iG network.
    g_mem->All_sync_count2[params[threadId].iL]++;
    //printf("Number of instances of full sync = %d \n",All_sync_count2[iL]);
    //fprintf(all_sync,"Number of instances of full sync = %d \n",All_sync_count2[0]);
  }

  // TOASK: What is happening here?
  // Write spike time on file
  for(kk=0;kk<N;kk++) {
    tmp1 = 10000*results[threadId].tspike[kk][spike_count[kk]-7];
    tmp2 = 10000*results[threadId].tspike[kk][spike_count[kk]-8];
    tmp3 = 10000*results[threadId].tspike[kk][spike_count[kk]-9];
    tmp4 = 10000*results[threadId].tspike[kk][spike_count[kk]-10];
    tmp5 = 10000*results[threadId].tspike[kk][spike_count[kk]-11];
    tmp6 = 10000*results[threadId].tspike[kk][spike_count[kk]-12];
    tmp7 = 10000*results[threadId].tspike[kk][spike_count[kk]-13];
  //fprintf(spike_time,"%d \t %lu \t %lu \t %lu \t %lu \t %lu \t \%d \n",kk,tmp1,tmp2,tmp3,tmp4,tmp5,flag_unconnctd_graph);
                      //fprintf(spike_time,"%d \t %lu \t %lu \t %lu \t %lu \t %lu \t %lu \t %lu \n",kk,tmp1,tmp2,tmp3,tmp4,tmp5,tmp6,tmp7);
  }
}


int main() {
  int num_simulations = (NL_max - NL_min) / NL_step * Ng_max * Nic;
  int tmax = 20;

  // Initialize the weight matrices in the GPU memory
  void *d_w;
  hipMalloc(&d_w, (NL_max - NL_min) * Ng_max / NL_step * N * N * sizeof(double));
  store_weights<<<1, (NL_max - NL_min) / NL_step>>>(d_w);

  // Initialize the global GPU memory
  global_mem g_mem;
  global_mem *d_g_mem;
  hipMalloc(&d_g_mem, sizeof(global_mem));
  g_mem.dt = 0.0002;
  g_mem.epsilon = 0.01;
  g_mem.vth = 0.8;
  g_mem.vreset = 0;
  g_mem.a = 1;
  g_mem.b = 1;
  g_mem.tol = 0.0001;
  g_mem.Nstep = tmax / g_mem.dt;
  for(int i = 0; i < NL_max - NL_min + 1; ++i) {
    for(int j = 0; j < Ng_max; ++j) {
      g_mem.All_sync_count1[i][j] = 0;
    }
    g_mem.All_sync_count2[i] = 0;
  }
  hipMemcpy(d_g_mem, &g_mem, sizeof(g_mem), hipMemcpyHostToDevice);

  // Initialize the parameters
  simulation_params params[num_simulations];
  simulation_params *d_params;
  hipMalloc(&d_params, sizeof(params));
  for(int iL = 0; iL <= (NL_max - NL_min) / NL_step; ++iL) {
    for(int ig = 0; ig < Ng_max; ++ig) {
      for(int ic = 0; ic < Nic; ++ic) {
        params[iL][ig][ic].iL = iL;
        params[iL][ig][ic].ig = ig;
        params[iL][ig][ic].ic = ic;
      }
    }
  }
  hipMemcpy(d_params, params, sizeof(params), hipMemcpyHostToDevice);

  // Perform checks (debug)
  check_weights<<<1, 1>>>(d_params);
  check_g_mem<<<1, 1>>>(d_g_mem);
  check_params<<<1, 1>>>(d_params);

  // Allocate memory for storing results
  simulation_result results[num_simulations];
  simulation_result *d_results;
  hipMalloc(&d_results, sizeof(results));
  // Start all simulations simultaneously
  simulate<<<1, num_simulations>>>(d_params, d_results, d_g_mem, d_w);

  return 0;
}

/* Check the weights on GPU memory */
__global__ void check_weights(double w[(NL_max - NL_min) * Ng_max / NL_step][N][N]) {
  int n = (NL_max - NL_min) * Ng_max / NL_step;
  for(int i = 0; i < n; ++i) {
    printf("\nnL = %d\tng = %d\n\t", i / Ng_max, i % Ng_max);
    for(int j = 0; j < N; ++j) {
      for(int k = 0; k < N; ++k) {
        printf("%.0lf ");
      }
      printf("\n\t");
    }
  }
}

/* Check the global data on GPU memory */
__global__ void check_g_mem(global_mem *g_mem) {
  printf("dt = %lf\n", dt);
  printf("epsilon = %lf\n", epsilon);
  printf("vth = %lf\n", vth);
  printf("vreset = %lf\n", vreset);
  printf("a = %lf\n", a);
  printf("b = %lf\n", b);
  printf("tol = %lf\n", tol);
  printf("\n");
}

/* Check the simulation parameters on GPU memory */
__global__ void check_params(simulation_params *params) {
  int n = (NL_max - NL_min) * Ng_max * Nic / NL_step;
  for(int i = 0; i < n; ++i) {
    printf("iL = %d\t", params[i].iL);
    printf("ig = %d\t", params[i].ig);
    printf("ic = %d\n", params[i].ic);
  }
  printf("\n");
}